#include "hip/hip_runtime.h"
/*

GPU Implementation of K-Means Algorithm

*/

#ifdef _MSC_VER // Compiling under Visual Studio
#include <Windows.h>

#include "hip/hip_runtime.h"
#include ""

#endif

// this code is for GPU kernel
#include <time.h>
#include "KMeansGPU.h"

using namespace KMG;

// GPU FUNCTION
__global__ void gpu(
	float *icdMat,
	int *ridMat,
	float *centers,
	float *points,
	int *label,
	int k,
	int dim)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int oldCnt = label[idx];
	float oldDist = 0;
	int i, j, curCnt, newCnt = oldCnt;
	float curDist, newDist;
	for(i = 0; i < dim; i++)
	{
		oldDist += (points[(idx * dim)+i] - centers[(oldCnt * dim)+i]) * (points[(idx * dim)+i] - centers[(oldCnt * dim)+i]);
	}
	oldDist = sqrt(oldDist);
	newDist = oldDist;

	for(j=0; j < k; j++){
		curCnt = ridMat[oldCnt*k+j];
		if(icdMat[oldCnt * k + curCnt] <= (2 * oldDist)){
			curDist = 0;
			for(i = 0; i < dim; i++)
			{
				curDist += (points[(idx * dim) + i] - centers[(curCnt * dim)+i]) * (points[(idx * dim)+i] - centers[(curCnt * dim)+i]);
			}
			curDist = sqrt(curDist);
			if(curDist < newDist){
				newDist = curDist;
				newCnt = curCnt;
			}
		} else {
			j=k;
		}
	}
	label[idx]=newCnt;
}

// FUNCTION CALLED BY MAIN
int gpu_kernel(KMeansGPU& kmeansGpu){
	const int dim = kmeansGpu.data.getDim();
	Centers& curr = kmeansGpu.curr;
	// k= number of clusters
	const int k = curr.nCenters;
	// int *d_k;
	// INTER CENTROID DISTANCES
	float* icdMat = curr.distMat; // this pointer is for CPU
	float *h_icdMat;
	float* d_icdMat; // this pointer is for GPU

	// RANKED INDEX MATRIX
	const int* ridMat = curr.ridMat;
	const int* h_ridMat;
	int* d_ridMat; // this pointer is for GPU
	
	Point * centers = curr.centers; // this centers is for CPU
	float *d_centers; // this centers is for GPU
	Point * points = kmeansGpu.data.getPoints();
	float *d_points; // this is for GPU
	const int N = kmeansGpu.data.getNPoints(); 

	float * h_centers = new float[kmeansGpu.nCenters * dim];
	float *h_points;	
	float total_entropy ;
	int* label = kmeansGpu.clusterIdx; // curr status of label in CPU this need to be updated after an iteration in GPU
	int *d_label;
	float calculate_entropy(float *, int *, float *, int , int );
	// getchar();
#ifdef _MSC_VER // Compiling under Visual Studio
	unsigned long startTime_, diffInMilliSec;
	startTime_ = GetTickCount();
#else
	struct timespec t_start, t_end;
	double elapsedTime;
	clock_gettime(CLOCK_REALTIME, &t_start);
#endif
	h_points = (float *)malloc(N*dim*sizeof(float));
	/* Allocate Memory on device */
	if ( hipSuccess != hipMalloc((void **)&d_icdMat, k*k*sizeof(float))) {printf( "Error!\n" ); exit(EXIT_FAILURE);}
	if ( hipSuccess != hipMalloc((void **)&d_ridMat, k*k*sizeof(int))) {printf( "Error!\n" ); exit(EXIT_FAILURE);}
	if ( hipSuccess != hipMalloc((void **)&d_centers, k*dim*sizeof(float))) {printf( "Error!\n" ); exit(EXIT_FAILURE);}
	if ( hipSuccess != hipMalloc((void **)&d_points, N*dim*sizeof(float))) {printf( "Error!\n" ); exit(EXIT_FAILURE);}
	if ( hipSuccess != hipMalloc((void **)&d_label, N*sizeof(int))) {printf( "Error!\n" ); exit(EXIT_FAILURE);}

#ifdef _MSC_VER // Compiling under Visual Studio
	diffInMilliSec = GetTickCount() - startTime_;
	printf("Time taken for memory allocation on CUDA device : %10lu millisec\n", diffInMilliSec);
#else
	clock_gettime(CLOCK_REALTIME, &t_end);
	elapsedTime = (t_end.tv_sec - t_start.tv_sec)*1000.0;
	elapsedTime += (t_end.tv_nsec - t_start.tv_nsec)/1000000.0;
	printf("time elapsed New Method (Memory Allocation) %1f msec\n", elapsedTime);
#endif

	int i,j;
	int nStages = kmeansGpu.nStages, iteration;
	curr.updateDistMat();
	curr.updateRidMat();

	printf("Number of Centers: %d Dimension: %d\n",k,dim);
#ifdef LOG
	for(i=0;i<N;i++){
		printf("--- label[%d] %d   ---\n",i,label[i]);
	}	
#endif
	for(i=0;i<k;i++){
		for(j=0;j<dim;j++){
			h_centers[i * dim + j] = centers[i].x[j];
			// printf("--- h_centers[%d][%d]= %f   ---\n",i,j,h_centers[i][j]);
		}
	}
	for(i=0;i<N;i++){
		for(j=0;j<dim;j++){
			h_points[i*dim+j] = points[i].x[j];
			// printf("--- h_points[%d][%d]= %f    ---\n",i,j,h_points[i][j]);
		}
	}
	h_ridMat = ridMat;
	h_icdMat = icdMat;
	
	// printf("size of int=%d  size of float %d\n", sizeof(int), sizeof(float));
#ifdef _MSC_VER // Compiling under Visual Studio
	startTime_ = GetTickCount();
#else
	clock_gettime(CLOCK_REALTIME, &t_start);
#endif
	hipMemcpy(d_points, (float *)h_points , N * dim * sizeof(float) , hipMemcpyHostToDevice);

#ifdef _MSC_VER // Not compiling under Visual Studio	
	diffInMilliSec = GetTickCount() - startTime_;
	printf("Time taken for memory copy on CUDA device : %10lu millisec\n", diffInMilliSec);

	startTime_ = GetTickCount();
#else
	clock_gettime(CLOCK_REALTIME, &t_end);
	elapsedTime = (t_end.tv_sec - t_start.tv_sec)*1000.0;
	elapsedTime += (t_end.tv_nsec - t_start.tv_nsec)/1000000.0;
	printf("time elapsed New Method (MemCpy outside for loop) %1f msec\n", elapsedTime);
	clock_gettime(CLOCK_REALTIME, &t_start);
#endif	

	/* This assertion ensures that the data set is in order. */
	assert (N % THREADS_PER_BLOCK == 0);
	
	for(iteration = 0 ; iteration < nStages ; iteration++) { 
		printf( "Stage %2d starting..", iteration + 1);

		/* Memory copy from HOST to GPU */
		hipMemcpy(d_icdMat, (float *)h_icdMat, k*k*sizeof(float) , hipMemcpyHostToDevice);
		hipMemcpy(d_ridMat, (int *)h_ridMat, k*k*sizeof(int) , hipMemcpyHostToDevice);
		hipMemcpy(d_label, label , N*sizeof(int) , hipMemcpyHostToDevice);
		hipMemcpy(d_centers, (float *)h_centers ,k*dim*sizeof(float) , hipMemcpyHostToDevice);

		/* Main Computation call on GPU */
		gpu<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>> (d_icdMat, d_ridMat, d_centers, d_points, d_label, k, dim);

		/* Memory copy from GPU to HOST */
		hipMemcpy(label, d_label, N*sizeof(int), hipMemcpyDeviceToHost);

		/* Update centers from the new assignments */
		// for(int q = 0; q < N; q++, printf("%3d ", label[q]));printf("\n");

		curr.resetPerIteration();
		curr.processLabel(label, kmeansGpu.data);

		curr.updateCenters(true, h_centers);	
		curr.updateDistMat();
		curr.updateRidMat();

		// curr.print();
		printf( " done.\n");
	}
#ifdef _MSC_VER // Compiling under Visual Studio
	diffInMilliSec = GetTickCount() - startTime_;
	printf("Time taken for total computation on CUDA device : %10lu millisec\n", diffInMilliSec);
#else
	clock_gettime(CLOCK_REALTIME, &t_end);
	elapsedTime = (t_end.tv_sec - t_start.tv_sec)*1000.0;
	elapsedTime += (t_end.tv_nsec - t_start.tv_nsec)/1000000.0;
	printf("time elapsed New Method (iteration: after the loop) %1f msec\n", elapsedTime);
#endif

#ifdef LOG
	printf("Printing label[i]s\n");
	for(i=0;i<N;i++){
		printf(" %6d",label[i]);
	}
#endif
	
	hipFree(d_points); hipFree(d_centers);
	hipFree(d_icdMat); hipFree(d_ridMat); hipFree(d_label);
	
	total_entropy = calculate_entropy(h_centers, label, h_points, N, dim);
	printf("\n total entropy of the system = %f\n", total_entropy);
	if(!h_centers) delete [] h_centers;
	return 0;
}

float calculate_entropy(float *h_centers, int *label, float *h_points, int N, int dim){
	int i,j,center_idx;
	float total=0;
	for(i=0;i<N;i++){
		center_idx = label[i];
		for(j=0;j<dim;j++){
	total += ((h_centers[center_idx*dim+j]-h_points[i*dim+j])*(h_centers[center_idx*dim+j]-h_points[i*dim+j]));
		}
	}
	return sqrt(total)/N;
}
